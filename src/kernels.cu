#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/distance.h>
#include <thrust/copy.h>
#include <thrust/tuple.h>
#include <thrust/fill.h>
#include <kernels/defaults.h>
#include <kernels/kernels.h>
#include <utils/exceptions.hpp>
#include <utils/utils.hpp>
#include <thrust/adjacent_difference.h>
#include <math.h>
#include <thrust/system/cuda/vector.h>
#include <thrust/system/cuda/execution_policy.h>
#include <map>
#include <typeinfo>

#define SQRT2 1.4142135623730951f

//--------------Harmonic summing----------------//

/* Unwrapped for 3x speed increase */
__global__
void harmonic_sum_kernel(float *d_idata, float **d_odata,
			 size_t size, unsigned nharms)
  
{
  for( int idx = blockIdx.x*blockDim.x + threadIdx.x ; idx < size ; idx += blockDim.x*gridDim.x )
    {
      float val = d_idata[idx];
      
      if (nharms>0)
	{
      	  val += d_idata[(int) (idx*0.5 + 0.5)];
	  d_odata[0][idx] = val*rsqrt(2.0);
	}
      
      if (nharms>1)
	{
	  val += d_idata[(int) (idx * 0.75 + 0.5)];
	  val += d_idata[(int) (idx * 0.25 + 0.5)];
	  d_odata[1][idx] = val*0.5;
	}

      if (nharms>2)
	{
	  val += d_idata[(int) (idx * 0.125 + 0.5)];
	  val += d_idata[(int) (idx * 0.375 + 0.5)];
	  val += d_idata[(int) (idx * 0.625 + 0.5)];
	  val += d_idata[(int) (idx * 0.875 + 0.5)];
	  d_odata[2][idx] = val*rsqrt(8.0);
	}

      if (nharms>3)
	{
	  val += d_idata[(int) (idx * 0.0625 + 0.5)];
	  val += d_idata[(int) (idx * 0.1875 + 0.5)];
	  val += d_idata[(int) (idx * 0.3125 + 0.5)];
	  val += d_idata[(int) (idx * 0.4375 + 0.5)];
	  val += d_idata[(int) (idx * 0.5625 + 0.5)];
	  val += d_idata[(int) (idx * 0.6875 + 0.5)];
	  val += d_idata[(int) (idx * 0.8125 + 0.5)];
	  val += d_idata[(int) (idx * 0.9375 + 0.5)];
	  d_odata[3][idx] = val*0.25;
	}
      
      if (nharms>4)
	{
	  val += d_idata[(int) (idx * 0.03125 + 0.5)];
	  val += d_idata[(int) (idx * 0.09375 + 0.5)];
	  val += d_idata[(int) (idx * 0.15625 + 0.5)];
	  val += d_idata[(int) (idx * 0.21875 + 0.5)];
	  val += d_idata[(int) (idx * 0.28125 + 0.5)];
	  val += d_idata[(int) (idx * 0.34375 + 0.5)];
	  val += d_idata[(int) (idx * 0.40625 + 0.5)];
	  val += d_idata[(int) (idx * 0.46875 + 0.5)];
	  val += d_idata[(int) (idx * 0.53125 + 0.5)];
	  val += d_idata[(int) (idx * 0.59375 + 0.5)];
	  val += d_idata[(int) (idx * 0.65625 + 0.5)];
	  val += d_idata[(int) (idx * 0.71875 + 0.5)];
	  val += d_idata[(int) (idx * 0.78125 + 0.5)];
	  val += d_idata[(int) (idx * 0.84375 + 0.5)];
	  val += d_idata[(int) (idx * 0.90625 + 0.5)];
	  val += d_idata[(int) (idx * 0.96875 + 0.5)];
	  d_odata[4][idx] = val*rsqrt(32.0);
	}
    }
  return;
}

/*
__global__
void harmonic_sum_kernel_wshared(float *d_idata, float **d_odata,
                         size_t size, unsigned nharms)

{
  
  __shared__ float buffer [sizeof(float)*512];
    

  for( int idx = blockIdx.x*blockDim.x + threadIdx.x ; idx < size ; idx += blockDim.x*gridDim.x )
    {
      float val = d_idata[idx];

      int thread_by_fold;
      int blockdim_by_fold;

      if (nharms>0)
        {
	  
	  thread_by_fold = threadIdx.x/2;
	  if (threadIdx.x % 2 == 0)
	    {
	      buffer[thread_by_fold] = d_idata[(int) (idx*0.5)];
	    }
	  //__syncthreads();
	  
	  val += buffer[thread_by_fold];
          d_odata[0][idx] = val*rsqrt(2.0);
        }

      if (nharms>1)
        {
	  thread_by_fold = threadIdx.x/4;
	  blockdim_by_fold = blockDim.x/4;
	  if (threadIdx.x % 4 == 0)
            {
              buffer[thread_by_fold]                    = d_idata[(int) (idx*0.75)];
	      buffer[thread_by_fold + blockdim_by_fold] = d_idata[(int) (idx*0.25)];
            }
          //__syncthreads();
	  val += buffer[thread_by_fold];
	  val += buffer[thread_by_fold + blockdim_by_fold];
	  d_odata[1][idx] = val*0.5;
        }

      if (nharms>2)
        {
	  thread_by_fold = threadIdx.x/8;
          blockdim_by_fold = blockDim.x/8;
	  if (threadIdx.x % 8 == 0)
            {
              buffer[thread_by_fold]                     = d_idata[(int) (idx*0.125)];
              buffer[thread_by_fold+ blockdim_by_fold]   = d_idata[(int) (idx*0.375)];
	      buffer[thread_by_fold+ 2*blockdim_by_fold] = d_idata[(int) (idx*0.625)];
	      buffer[thread_by_fold+ 3*blockdim_by_fold] = d_idata[(int) (idx*0.875)];
            }
          //__syncthreads();

	  val += buffer[thread_by_fold];
	  val += buffer[thread_by_fold + blockdim_by_fold];
	  val += buffer[thread_by_fold + 2*blockdim_by_fold];
	  val += buffer[thread_by_fold + 3*blockdim_by_fold];
          d_odata[2][idx] = val*rsqrt(8.0);
        }

      if (nharms>3)
        {
	  thread_by_fold = threadIdx.x/16;
          blockdim_by_fold = blockDim.x/16;
          if (threadIdx.x % 16 == 0)
            {
              buffer[thread_by_fold]                     = d_idata[(int) (idx*0.0625)];
              buffer[thread_by_fold+ blockdim_by_fold]   = d_idata[(int) (idx*0.1875)];
              buffer[thread_by_fold+ 2*blockdim_by_fold] = d_idata[(int) (idx*0.3125)];
              buffer[thread_by_fold+ 3*blockdim_by_fold] = d_idata[(int) (idx*0.4375)];
	      buffer[thread_by_fold+ 4*blockdim_by_fold] = d_idata[(int) (idx*0.5625)];
	      buffer[thread_by_fold+ 5*blockdim_by_fold] = d_idata[(int) (idx*0.6875)];
	      buffer[thread_by_fold+ 6*blockdim_by_fold] = d_idata[(int) (idx*0.8125)];
	      buffer[thread_by_fold+ 7*blockdim_by_fold] = d_idata[(int) (idx*0.9375)];
            }
          //__syncthreads();
	  
	  val += buffer[thread_by_fold];
	  val += buffer[thread_by_fold+ blockdim_by_fold];
	  val += buffer[thread_by_fold+ 2*blockdim_by_fold];
	  val += buffer[thread_by_fold+ 3*blockdim_by_fold];
	  val += buffer[thread_by_fold+ 4*blockdim_by_fold];
	  val += buffer[thread_by_fold+ 5*blockdim_by_fold];
	  val += buffer[thread_by_fold+ 6*blockdim_by_fold];
	  val += buffer[thread_by_fold+ 7*blockdim_by_fold];
	  
          d_odata[3][idx] = val*0.25;
        }
    }
  return;
  }*/

void device_harmonic_sum(float* d_input_array, float** d_output_array,
			 size_t size, unsigned nharms, 
			 unsigned int max_blocks, unsigned int max_threads)
{
  unsigned blocks = size/max_threads + 1;
  if (blocks > max_blocks)
    blocks = max_blocks;
  harmonic_sum_kernel<<<blocks,max_threads>>>(d_input_array,d_output_array,size,nharms);
  ErrorChecker::check_cuda_error("Error from device_harmonic_sum");
}

//------------spectrum forming--------------//


//Could be optimised with shared memory

__global__ 
void power_series_kernel(hipfftComplex *d_idata, float* d_odata, 
			 size_t size, size_t gulp_index)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_index;
  hipfftComplex& x = d_idata[idx];
  if(idx<size)
    {
      float z = x.x*x.x+x.y*x.y;
      d_odata[idx] = z*rsqrtf(z);
    }
  return;
}

//Could be optimised with shared memory

__global__ void bin_interbin_series_kernel(hipfftComplex *d_idata,float* d_odata, 
					   size_t size, size_t gulp_index)
{
  float* d_idata_float = (float*)d_idata;
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_index;
  float re_l =0.0;
  float im_l =0.0;
  if (idx>0 && idx<size) {
    re_l = d_idata_float[2*idx-2];
    im_l = d_idata_float[2*idx-1];
  }
  if(idx<size)
    {
      float re = d_idata_float[2*idx];
      float im = d_idata_float[2*idx+1];
      float ampsq = re*re+im*im;
      float ampsq_diff = 0.5*((re-re_l)*(re-re_l) +
                              (im-im_l)*(im-im_l));
      d_odata[idx] = sqrtf(fmaxf(ampsq,ampsq_diff));
    }
  return;
}

 /*
__global__ void bin_interbin_series_kernel(hipfftComplex *d_idata,float* d_odata, int size)
{
  int idx = blockIdx.x * (blockDim.x-1) + threadIdx.x;
  
  if (idx>=size-1)
    return;
    
  extern __shared__ hipfftComplex s[];

  //blockIdx accounts for backshift by 1 sample to keep single write coalesence
  
  if (idx!=0)
    s[threadIdx.x] = d_idata[idx-1];
  else
    s[threadIdx.x] = make_hipComplex(0.0,0.0);
  __syncthreads();
  
  if (threadIdx.x+1 == blockDim.x)
    return;

  hipfftComplex x = s[threadIdx.x+1];
  hipfftComplex y = s[threadIdx.x];
  float ampsq = x.x*x.x+x.y*x.y;
  float ampsq_diff = 0.5*((x.x-y.x)*(x.x-y.x) +
			  (x.y-y.y)*(x.y-y.y));
  float val = max(ampsq,ampsq_diff);
  d_odata[idx] = val*rsqrtf(val);
  
  return;
}
 */

void device_form_power_series(hipfftComplex* d_array_in, 
			      float* d_array_out,
			      size_t size, int way,
			      unsigned int max_blocks,
			      unsigned int max_threads)
{
  BlockCalculator calc(size,max_blocks,max_threads);
  for (int ii=0;ii<calc.size();ii++){
    if (way == 1)  
      bin_interbin_series_kernel<<<calc[ii].blocks,max_threads>>>
        (d_array_in, d_array_out, size, calc[ii].data_idx);
    else
      power_series_kernel<<<calc[ii].blocks,max_threads>>>
	(d_array_in, d_array_out, size, calc[ii].data_idx);
  }
  ErrorChecker::check_cuda_error("Error from device_form_power_series");
  return;
}

//-----------------time domain resampling---------------//

inline __device__ unsigned long getAcceleratedIndex(double accel_fact, double size_by_2,
						    unsigned long id){
  return __double2ull_rn(id + accel_fact*( ((id-size_by_2)*(id-size_by_2)) - (size_by_2*size_by_2)));
}


inline __device__ unsigned long getAcceleratedIndexII(double accel_fact, double size,
						      unsigned long id){
  return __double2ull_rn(id + id*accel_fact*(id-size));
}

//inline __device__ unsigned long getAcceleratedIndexIII(double accel_fact, double jerk_fact,double size,
//						      unsigned long id){
 // return __double2ull_rn(id + id*accel_fact*(id-size) + id*id*jerk_fact*(id-size));
//}

inline __device__ unsigned long getAcceleratedIndexIII(double accel_fact, double jerk_fact,double size,
						      unsigned long id){
  return __double2ull_rn(id + id*accel_fact*(id-size) + (size/2.0-id)*(size/2.0-id)*jerk_fact*(size/2.0-id));
}


/*inline __device__ unsigned long getAcceleratedIndexIV(double accel_fact, double jerk_fact,double size,
						      unsigned long id){
  signed long value = __double2ll_rn(id + id*accel_fact*(id-size) + (size/2.0-id)*(size/2.0-id)*jerk_fact*(size/2.0-id));

  if(value < 0)
   {
    printf("%lld\n",value);
    //printf("Here!");
    value = 0;
   }
  else if (value > size-1)
      value = size-1;
  else
      value=value;
  unsigned long new_value = value;
  return new_value; 
}
*/


__global__ void resample_kernel(float* input_d,
				float* output_d,
				double accel_fact,
				size_t size,
				double size_by_2,
				size_t start_idx)
{
  unsigned long idx = threadIdx.x + blockIdx.x * blockDim.x + start_idx;
  if (idx>=size)
    return;
  unsigned long idx_read = getAcceleratedIndex(accel_fact,size_by_2,idx);
  output_d[idx] = input_d[idx_read];
}


__global__ void resample_kernelII(float* input_d,
				  float* output_d,
				  double accel_fact,
				  double size)
				  
{
  for( unsigned long idx = blockIdx.x*blockDim.x + threadIdx.x ; idx < size ; idx += blockDim.x*gridDim.x )
  {
    unsigned long out_idx = getAcceleratedIndexII(accel_fact,size,idx);
    output_d[idx] = input_d[out_idx];
  }
}

__global__ void resample_kernelIII(float* input_d,
				  float* output_d,
				  double accel_fact,
                                  double jerk_fact, 
				  double size)
				  
{
   
  //std::cout << typeid(size).name() << "\n";
  //printf("size is %f",size);
  for( unsigned long idx = blockIdx.x*blockDim.x + threadIdx.x ; idx < size ; idx += blockDim.x*gridDim.x )
  {
    unsigned long out_idx;
    unsigned long cnt1=0;
    //if(jerk_fact < 0.0)  
    //    out_idx = getAcceleratedIndexIV(accel_fact,jerk_fact,size,idx);
    //else

    out_idx = getAcceleratedIndexIII(accel_fact,jerk_fact,size,idx);
    if (out_idx > size -1)
    {
        cnt1++;
        output_d[idx] = 0.0; // pad with zero
    }
   //if (out_idx > size -1)
   //      cnt2++;       
   //printf("%lu %lu \n",idx,out_idx);
    else
    {
        output_d[idx] = input_d[out_idx];
    }
  }
}






void device_resampleII(float * d_idata, float * d_odata,
                     size_t size, float a,
                     float tsamp, unsigned int max_threads,
                     unsigned int max_blocks)
{
  
  double accel_fact = ((a*tsamp) / (2 * 299792458.0));
  unsigned blocks = size/max_threads + 1;
  if (blocks > max_blocks)
    blocks = max_blocks;
  resample_kernelII<<< blocks,max_threads >>>(d_idata, d_odata,
					      accel_fact,
					      (double) size);
  ErrorChecker::check_cuda_error("Error from device_resampleII");
}

void device_resampleIII(float * d_idata, float * d_odata,
                     size_t size, float a, float j,
                     float tsamp, unsigned int max_threads,
                     unsigned int max_blocks)
{
  
  double accel_fact = ((a*tsamp) / (2 * 299792458.0));
  //printf("tsamp: %.*f\n",20,tsamp);
  double jerk_fact = ((j*tsamp*tsamp) / (6 * 299792458.0));
  //double size_by_2  = (double)size/2.0;

  unsigned blocks = size/max_threads + 1;
  if (blocks > max_blocks)
    blocks = max_blocks;
  resample_kernelIII<<< blocks,max_threads >>>(d_idata, d_odata,
					      accel_fact, jerk_fact,
					      (double) size);
  ErrorChecker::check_cuda_error("Error from device_resampleIII");
}

void device_resample(float * d_idata, float * d_odata,
		     size_t size, float a, 
		     float tsamp, unsigned int max_threads,
		     unsigned int max_blocks)
{
  double accel_fact = ((a*tsamp) / (2 * 299792458.0));
  double size_by_2  = (double)size/2.0;
  BlockCalculator calc(size,max_blocks,max_threads);
  for (int ii=0;ii<calc.size();ii++)
    resample_kernel<<< calc[ii].blocks,max_threads >>>(d_idata, d_odata, 
						       accel_fact,
						       size,
						       size_by_2,
						       calc[ii].data_idx);
  ErrorChecker::check_cuda_error("Error from device_resample");
}

//------------------peak finding-----------------//
//defined here as (although Thrust based) requires CUDA functors

struct greater_than_threshold : thrust::unary_function<thrust::tuple<int,float>,bool>
{
  float threshold;
  __device__ bool operator()(thrust::tuple<int,float> t) { return thrust::get<1>(t) > threshold; }
  greater_than_threshold(float thresh):threshold(thresh){}
};

int device_find_peaks(int n, int start_index, float * d_dat,
		      float thresh, int * indexes, float * snrs,
		      thrust::device_vector<int>& d_index, 
		      thrust::device_vector<float>& d_snrs,
		      cached_allocator& policy)
{
  
  using thrust::tuple;
  using thrust::counting_iterator;
  using thrust::zip_iterator;
  // Wrap the device pointer to let Thrust know                              
  thrust::device_ptr<float> dptr_dat(d_dat + start_index);
  typedef thrust::device_vector<float>::iterator snr_iterator;
  typedef thrust::device_vector<int>::iterator indices_iterator;
  thrust::counting_iterator<int> iter(start_index);
  zip_iterator<tuple<counting_iterator<int>,thrust::device_ptr<float> > > zipped_iter = make_zip_iterator(make_tuple(iter,dptr_dat));
  zip_iterator<tuple<indices_iterator,snr_iterator> > zipped_out_iter = make_zip_iterator(make_tuple(d_index.begin(),d_snrs.begin()));
  
  //apply execution policy to get some speed up
  int num_copied = thrust::copy_if(thrust::cuda::par(policy), zipped_iter, zipped_iter+n-start_index,
				   zipped_out_iter,greater_than_threshold(thresh)) - zipped_out_iter;
  thrust::copy(d_index.begin(),d_index.begin()+num_copied,indexes);
  thrust::copy(d_snrs.begin(),d_snrs.begin()+num_copied,snrs);
  ErrorChecker::check_cuda_error("Error from device_find_peaks;");
  return(num_copied);
}

//------------------rednoise----------------//

template<typename T>
struct square {
    __host__ __device__ inline
    T operator()(const T& x) { return x*x; }
};

template<typename T>
float GPU_rms(T* d_collection,int nsamps, int min_bin)
{
  T rms_sum;
  float rms;

  using thrust::device_ptr;
  rms_sum = thrust::transform_reduce(device_ptr<T>(d_collection)+min_bin,
				     device_ptr<T>(d_collection)+nsamps,
				     square<T>(),T(0),thrust::plus<T>());
  rms = sqrt(float(rms_sum)/float(nsamps-min_bin));
  return rms;
}

template<typename T>
float GPU_mean(T* d_collection,int nsamps, int min_bin)
{
  float mean;
  T m_sum;

  using thrust::device_ptr;
  m_sum = thrust::reduce(device_ptr<T>(d_collection)+min_bin,
			 device_ptr<T>(d_collection)+nsamps);

  hipDeviceSynchronize();
  mean = float(m_sum)/float(nsamps-min_bin);

  return mean;
}


template<typename T>
void GPU_fill(T* start, T* end, T value){
  thrust::device_ptr<T> ar_start(start);
  thrust::device_ptr<T> ar_end(end);
  thrust::fill(ar_start,ar_end,value);
  ErrorChecker::check_cuda_error("Error in GPU_fill");
}

template void GPU_fill<float>(float*, float*, float);
template float GPU_rms<float>(float*,int,int);
template float GPU_mean<float>(float*,int,int);

__global__
void normalisation_kernel(float*d_powers, float mean, float sigma, 
			  size_t size, size_t gulp_idx)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_idx;
  if (idx>=size)
    return;
  float val = d_powers[idx];
  val-=mean;
  val/=sigma;
  d_powers[idx] = val;
}

void device_normalise(float* d_powers,
		      float mean,
		      float sigma,
		      unsigned int size,
		      unsigned int max_blocks,
		      unsigned int max_threads)
{
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++)
    normalisation_kernel<<<calc[ii].blocks,max_threads>>>(d_powers,mean,sigma,size,
							  calc[ii].data_idx);
  ErrorChecker::check_cuda_error("Error from device_normalise");
}


//old normalisation routine used after a different
//rednoise algorithm was applied
void device_normalise_spectrum(int nsamp,
			       float* d_power_spectrum,
			       float* d_normalised_power_spectrum,
			       int min_bin,
			       float * sigma)
{
  float mean;
  float rms;
  float meansquares;
  
  if (*sigma==0.0) {
    mean = GPU_mean(d_power_spectrum,nsamp,min_bin);
    rms = GPU_rms(d_power_spectrum,nsamp,min_bin);
    meansquares = rms*rms;
    *sigma = sqrt(meansquares - (mean*mean));
  }
  
  thrust::transform(thrust::device_ptr<float>(d_power_spectrum),
                    thrust::device_ptr<float>(d_power_spectrum)+nsamp,
                    thrust::make_constant_iterator(*sigma),
                    thrust::device_ptr<float>(d_normalised_power_spectrum),
                    thrust::divides<float>());
  ErrorChecker::check_cuda_error("Error from device_normalise_spectrum");
}


//--------------Time series folder----------------//
/*
__global__
void fold_filterbank_kernel(float* input, float* output, unsigned* count,
			    unsigned nchans, float tsamp_by_period,
			    double accel_fact, unsigned nbins, 
			    float nrots_per_subint, unsigned nsamps,
			    unsigned offset)
{
  extern __shared__ peasoup_fold_plan plan [];
  
  unsigned first_samp;
  unsigned samp;
  float rotation;
  float int_part;
  float frac_part;
  unsigned in_idx_partial,in_idx;
  unsigned out_idx_partial,out_idx;
  
  //Start in time domain and calculate output 
  //phasebin an subint for each sample in the block

  first_samp = blockIdx.x*blockDim.x + offset;
  samp = first_samp + threadIdx.x;
  rotation = (samp + samp*accel_fact*(samp-nsamps))*tsamp_by_period;
  frac_part = modf(rotation,&int_part);
  plan[threadIdx.x].subint = __float2uint_rd(rotation/nrots_per_subint);
  plan[threadIdx.x].phasebin = __float2uint_rd(frac_part*nbins);
  
  //Sync and move to channel domain to preserve
  //memory bandwidth
  
  __sync_threads();
  
  for (jj=0; jj<blockDim.x; jj++)
    {
      in_idx_partial = (jj+first_samp)*nchans;
 
      //These are shared memory broadcasts
      out_idx_partial = nbins*nchans*plan[jj].subint + nchan*plan[jj].bin;

      for (ii=threadIdx.x; ii<nchans; ii+=blockDim.x)
	{
	  in_idx = in_idx_partial+ii;
	  out_idx = out_idx_partial+ii;
	  output[out_idx] += input[in_idx];
	  count[out_idx]++;
	}
    }
}


int device_fold_filterbank(float* input, float* output, unsigned* count, 
			   float tsamp, float period, float acceleration,
			   unsigned nsubints, unsigned nbins, unsigned nchans,
			   unsigned total_nsamps, unsigned nsamps, unsigned offset,
			   unsigned max_blocks, unsigned max_threads)
{
  
  float tobs = total_nsamps*tsamp;
  float nrots = tobs/period;
  float nrots_per_subint = nrots/nsubints;
  float tsamp_by_period = tsamp_by_period;
  double accel_fact = ((acceleration * tsamp) / (2 * 299792458.0));
  unsigned mem_size_bytes = nsamps*sizeof(peasoup_fold_plan);
  fold_filterbank_kernel<<<max_blocks,max_threads,mem_size_bytes>>>
    (input, output, count, nchans, tsamp_by_period, accel_fact, nbins,
     nrots_per_subint, nsamps, offset);
    
     }*/


__global__ 
void fold_time_series_kernel(float* input, float* output, 
			     size_t nsubints,
			     size_t nbins, size_t nsamps_per_subint,
			     double tsamp_by_period)
{
  extern __shared__ float block [];
  float* soutput = (float*) &block[0];
  int* count = (int*) &block[nbins];

  //one block per subint
  size_t data_idx = nsamps_per_subint*blockIdx.x + threadIdx.x;
  size_t ii,jj;
  int idx;
  
  if (threadIdx.x>nbins)
    return;

  //zero output shared memory
  for (ii=threadIdx.x; ii<nbins; ii+=blockDim.x){
    soutput[ii] = 0;
    count[ii] = 1;
  }
  //read all data for a subint
  double int_part,float_part;
  
  for (jj = data_idx; jj < (data_idx + nsamps_per_subint); jj += blockDim.x)
    {
      float_part = modf(jj*tsamp_by_period,&int_part);
      idx = __double2int_rd(float_part * nbins);
      atomicAdd(&soutput[idx], input[jj]); 
      atomicAdd(&count[idx], 1);
    }
  
  for (ii=threadIdx.x; ii<nbins; ii+=blockDim.x)
    output[blockIdx.x * nbins + ii] = soutput[ii]/count[ii];
}

void device_fold_timeseries(float* input, float* output,
			    size_t nsamps, size_t nsubints,
			    double period, double tsamp, int nbins,
			    size_t max_blocks, size_t max_threads)
{
  size_t nsamps_per_subint = nsamps/nsubints;
  double tsamp_by_period = tsamp/period;

  if (nbins*sizeof(float)*2>16384){
    ErrorChecker::throw_error("device_fold_timeseries: nbins must be less than 2048");
    return;
  }

  fold_time_series_kernel<<<nsubints,nbins,2*nbins*nsubints*sizeof(float)>>>
    (input,output,nsubints,nbins,nsamps_per_subint,tsamp_by_period);
  ErrorChecker::check_cuda_error("Error from device_fold_timeseries.");
}

//--------------FoldOptimiser------------//
  
__device__ inline hipComplex cuCexpf(hipComplex z)
{
  hipComplex res;
  float t = expf(z.x);
  sincosf(z.y, &res.y, &res.x);
  res.x *= t;
  res.y *= t;
  return res;
}

__global__
void shift_array_generator_kernel(hipComplex* shift_ar, unsigned int shift_ar_size,
				  unsigned int nbins, unsigned int nints,
				  unsigned int nshift, float* shifts,
				  float two_pi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= shift_ar_size)
    return;
  float subint = idx/nbins%nints;
  unsigned int shift_idx = idx/(nbins*nints);
  unsigned int bin = idx%nbins;
  float shift = subint/nints * shifts[shift_idx];
  float ramp = bin*two_pi/nbins;
  if (bin>nbins/2)
    ramp-=two_pi;
  hipComplex tmp1 = make_hipComplex(0.0,-1*ramp*shift);
  hipComplex tmp2 = cuCexpf(tmp1);
  shift_ar[idx] = tmp2;
}

__global__
void template_generator_kernel(hipComplex* templates, unsigned int nbins, unsigned int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>=size)
    return;
  unsigned int bin = idx%nbins;
  unsigned int template_idx = idx/nbins;
  float val = (bin<=template_idx);
  templates[idx] = make_hipComplex(val,0.0);
}

__global__
void multiply_by_shift_kernel(hipComplex* input, hipComplex* output,
			      hipComplex* shift_array, unsigned int nbins_by_nints,
			      unsigned int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>=size)
    return;
  unsigned int in_idx = idx%(nbins_by_nints);
  output[idx] = hipCmulf(input[in_idx],shift_array[idx]);
}

__global__
void collapse_subints_kernel(hipComplex* input, hipComplex* output, 
			     unsigned int nbins, unsigned int nints, 
			     unsigned int nbins_by_nints, unsigned int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>=size)
    return;
  unsigned int bin = idx%nbins;
  unsigned int fold = idx/nbins;
  unsigned int in_idx = (fold*nbins_by_nints)+bin;
  hipComplex val =  make_hipComplex(0.0,0.0);
  for (int ii=0;ii<nints;ii++)
    val = hipCaddf(val,input[in_idx+ii*nbins]);  
  output[idx] = val;
}

__global__
void multiply_by_template_kernel(hipComplex* input, hipComplex* output,
				 hipComplex* templates, unsigned int nbins,
				 unsigned int nshifts, unsigned int nbins_by_nshifts,
				 unsigned int size, unsigned int step)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>=size)
    return;
  unsigned int template_idx = idx/nbins_by_nshifts;
  unsigned int bin = idx%nbins;
  unsigned int shift = idx%nbins_by_nshifts;
  float width = (template_idx+1.0);
  hipComplex normalisation_factor = make_hipComplex(sqrtf(width),0.0);
  if (bin==0)
    output[idx] = make_hipComplex(0.0,0.0);
  else
    output[idx] = hipCdivf(hipCmulf(input[shift],templates[template_idx*nbins+bin]),normalisation_factor);
}

__global__
void cuCabsf_kernel(hipComplex* input, float* output, unsigned int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>=size)
    return;
  output[idx] = hipCabsf(input[idx]);
}

__global__
void real_to_complex_kernel(float* input, hipComplex* output, unsigned int size) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>=size)
    return;
  output[idx] = make_hipComplex(input[idx],0.0);
}

unsigned int device_argmax(float* input, unsigned int size)
{
  thrust::device_ptr<float> ptr(input);
  thrust::device_ptr<float> max_elem = thrust::max_element(ptr,ptr+size);
  ErrorChecker::check_cuda_error("Error from thrust::max_element in device_argmax");
  return thrust::distance(ptr,max_elem);
}

void device_real_to_complex(float* input, hipComplex* output, unsigned int size, 
			    unsigned int max_blocks, unsigned int max_threads)
{
  BlockCalculator calc(size,max_blocks,max_threads);
  for (int ii=0;ii<calc.size();ii++)
    real_to_complex_kernel<<<calc[ii].blocks,max_threads>>>(input,output,size);
  ErrorChecker::check_cuda_error("Error from device_real_to_complex");
  return;
}


void device_get_absolute_value(hipComplex* input, float* output, unsigned int size,
			       unsigned int max_blocks, unsigned int max_threads)
{
  BlockCalculator calc(size,max_blocks,max_threads);
  for (int ii=0;ii<calc.size();ii++)
    cuCabsf_kernel<<<calc[ii].blocks,max_threads>>>(input,output,size);
  ErrorChecker::check_cuda_error("Error from device_get_absolute_value");
  return;
}

void device_generate_shift_array(hipComplex* shifted_ar,
                                 unsigned int shifted_ar_size,
                                 unsigned int nbins, unsigned int nints,
                                 unsigned int nshift, float* shifts,
                                 unsigned int max_blocks, unsigned int max_threads)
{
  float two_pi = 2*3.14159265359;
  BlockCalculator calc(shifted_ar_size,max_blocks,max_threads);
  for (int ii=0;ii<calc.size();ii++)
    shift_array_generator_kernel<<<calc[ii].blocks,max_threads>>>(shifted_ar, shifted_ar_size, nbins,
								  nints, nshift, shifts, two_pi);
  ErrorChecker::check_cuda_error("Error from device_generate_shift_array");
  return;
}

void device_generate_template_array(hipComplex* templates, unsigned int nbins, 
				    unsigned int size, unsigned int max_blocks,
				    unsigned int max_threads)
{
  BlockCalculator calc(size,max_blocks,max_threads);
  for (int ii=0;ii<calc.size();ii++){
    template_generator_kernel<<<calc[ii].blocks,max_threads>>>(templates, nbins, size);
  }
  ErrorChecker::check_cuda_error("Error from device_generate_template_array");
  return;
}

void device_multiply_by_shift(hipComplex* input, hipComplex* output,
                              hipComplex* shift_array, unsigned int size,
			      unsigned int nbins_by_nints,
			      unsigned int max_blocks, unsigned int max_threads)
{
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++){
    multiply_by_shift_kernel<<<calc[ii].blocks,max_threads>>>(input,output,shift_array,
							      nbins_by_nints,size);
  }
  ErrorChecker::check_cuda_error("Error from device_multiply_by_shift");
  return;
}

void device_collapse_subints(hipComplex* input, hipComplex* output,
			     unsigned int nbins, unsigned int nints,
			     unsigned int size, unsigned int max_blocks, 
			     unsigned int max_threads)
{
  unsigned int nbins_by_nints = nbins*nints;
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++){
    collapse_subints_kernel<<<calc[ii].blocks,max_threads>>>(input,output,nbins,
							     nints,nbins_by_nints,size);
  }
  ErrorChecker::check_cuda_error("Error from device_collapse_subints");
  return;
}
  
void device_multiply_by_templates(hipComplex* input, hipComplex* output,
				  hipComplex* templates, unsigned int nbins,
				  unsigned int nshifts,
				  unsigned int size, unsigned int step,
				  unsigned int max_blocks, unsigned int max_threads)
{
  unsigned int nbins_by_nshifts = nbins*nshifts;
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++){
    multiply_by_template_kernel<<<calc[ii].blocks,max_threads>>>(input,output,templates,
								 nbins,nshifts,nbins_by_nshifts,
								 size,step);
  }
  ErrorChecker::check_cuda_error("Error from device_multiply_by_templates");
  return;
}

//--------------Rednoise stuff--------------//

//Ben Barsdells median scrunching algorithm from Heimdall
/*
  Note: The implementations of median3-5 here can be derived from
          'sorting networks'.
*/

inline __host__ __device__
float median3(float a, float b, float c) {
	return a < b ? b < c ? b
	                      : a < c ? c : a
	             : a < c ? a
	                     : b < c ? c : b;
}
inline __host__ __device__
float median4(float a, float b, float c, float d) {
	return a < c ? b < d ? a < b ? c < d ? 0.5f*(b+c) : 0.5f*(b+d)
	                             : c < d ? 0.5f*(a+c) : 0.5f*(a+d)
	                     : a < d ? c < b ? 0.5f*(d+c) : 0.5f*(b+d)
	                             : c < b ? 0.5f*(a+c) : 0.5f*(a+b)
	             : b < d ? c < b ? a < d ? 0.5f*(b+a) : 0.5f*(b+d)
	                             : a < d ? 0.5f*(a+c) : 0.5f*(c+d)
	                     : c < d ? a < b ? 0.5f*(d+a) : 0.5f*(b+d)
	                             : a < b ? 0.5f*(a+c) : 0.5f*(c+b);
}
inline __host__ __device__
float median5(float a, float b, float c, float d, float e) {
	// Note: This wicked code is by 'DRBlaise' and was found here:
	//         http://stackoverflow.com/a/2117018
	return b < a ? d < c ? b < d ? a < e ? a < d ? e < d ? e : d
                                                 : c < a ? c : a
                                         : e < d ? a < d ? a : d
                                                 : c < e ? c : e
                                 : c < e ? b < c ? a < c ? a : c
                                                 : e < b ? e : b
                                         : b < e ? a < e ? a : e
                                                 : c < b ? c : b
                         : b < c ? a < e ? a < c ? e < c ? e : c
                                                 : d < a ? d : a
                                         : e < c ? a < c ? a : c
                                                 : d < e ? d : e
                                 : d < e ? b < d ? a < d ? a : d
                                                 : e < b ? e : b
                                         : b < e ? a < e ? a : e
                                                 : d < b ? d : b
	         : d < c ? a < d ? b < e ? b < d ? e < d ? e : d
                                                 : c < b ? c : b
                                         : e < d ? b < d ? b : d
                                                 : c < e ? c : e
                                 : c < e ? a < c ? b < c ? b : c
                                                 : e < a ? e : a
                                         : a < e ? b < e ? b : e
                                                 : c < a ? c : a
                         : a < c ? b < e ? b < c ? e < c ? e : c
                                                 : d < b ? d : b
                                         : e < c ? b < c ? b : c
                                                 : d < e ? d : e
                                 : d < e ? a < d ? b < d ? b : d
                                                 : e < a ? e : a
                                         : a < e ? b < e ? b : e
	                                         : d < a ? d : a;
}

struct median_scrunch5_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	median_scrunch5_kernel(const hd_float* in_)
		: in(in_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_float a = in[5*i+0];
		hd_float b = in[5*i+1];
		hd_float c = in[5*i+2];
		hd_float d = in[5*i+3];
		hd_float e = in[5*i+4];
		return median5(a, b, c, d, e);
	}
};

hd_error median_scrunch5(const hd_float* d_in,
                         hd_size         count,
                         hd_float*       d_out)
{
	thrust::device_ptr<const hd_float> d_in_begin(d_in);
	thrust::device_ptr<hd_float>       d_out_begin(d_out);
	
	if( count == 1 ) {
		*d_out_begin = d_in_begin[0];
	}
	else if( count == 2 ) {
		*d_out_begin = 0.5f*(d_in_begin[0] + d_in_begin[1]);
	}
	else if( count == 3 ) {
		*d_out_begin = median3(d_in_begin[0],
		                       d_in_begin[1],
		                       d_in_begin[2]);
	}
	else if( count == 4 ) {
		*d_out_begin = median4(d_in_begin[0],
		                       d_in_begin[1],
		                       d_in_begin[2],
		                       d_in_begin[3]);
	}
	else {
		// Note: Truncating here is necessary
		hd_size out_count = count / 5;
		using thrust::make_counting_iterator;
		thrust::transform(make_counting_iterator<unsigned int>(0),
		                  make_counting_iterator<unsigned int>(out_count),
		                  d_out_begin,
		                  median_scrunch5_kernel(d_in));
	}
	return HD_NO_ERROR;
}

struct linear_stretch_functor
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	hd_float        step;
	linear_stretch_functor(const hd_float* in_,
	                       hd_size in_count, hd_size out_count)
		: in(in_), step(hd_float(in_count-1)/(out_count-1)) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_float     x = i * step;
		unsigned int j = x;
		return in[j] + ((x-j > 1e-5f) ? (x-j)*(in[j+1]-in[j]) : 0.f);
	}
};

hd_error linear_stretch(const hd_float* d_in,
                        hd_size         in_count,
                        hd_float*       d_out,
                        hd_size         out_count)
{
	using thrust::make_counting_iterator;
	thrust::device_ptr<hd_float> d_out_begin(d_out);
	
	thrust::transform(make_counting_iterator<unsigned int>(0),
	                  make_counting_iterator<unsigned int>(out_count),
	                  d_out_begin,
	                  linear_stretch_functor(d_in, in_count, out_count));
	return HD_NO_ERROR;
}

__global__ 
void divide_c_by_f_kernel(hipComplex* c, float* f, unsigned int size, unsigned int gulp_idx)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_idx;
  if (idx>=size)
    return;
  if (idx<5)
    c[idx] = make_hipComplex(0.0,0.0);
  else
    c[idx] = hipCdivf(c[idx],make_hipComplex(f[idx],0.0));
}

void device_divide_c_by_f(hipComplex* c, float* f, unsigned int size,
			    unsigned int max_blocks, unsigned int max_threads)
{
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++){
    divide_c_by_f_kernel<<<calc[ii].blocks,max_threads>>>(c,f,size,ii*max_threads*max_blocks);
  }
  ErrorChecker::check_cuda_error();
  return;
}

__global__
void zap_birdies_kernel(hipComplex* fseries, float* birdies, float* widths,
			float bin_width, unsigned int size,
			unsigned int fseries_size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>=size)
    return;
  int ii;
  float freq = birdies[idx];
  float width = widths[idx];
  int low_bin = __float2int_rd((freq-width)/bin_width);
  int high_bin = __float2int_ru((freq+width)/bin_width);
  
  if (low_bin<0)
    low_bin = 0;
  if (low_bin>=fseries_size)
    return;
  if (high_bin>=fseries_size)
    high_bin = fseries_size-1;
  for (ii=low_bin;ii<high_bin;ii++)
    fseries[ii] = make_hipComplex(1.0,0.0);
}

void device_zap_birdies(hipComplex* fseries, float* d_birdies, float* d_widths, float bin_width,
			unsigned int birdies_size, unsigned int fseries_size,
			unsigned int max_blocks, unsigned int max_threads)
{
  BlockCalculator calc(birdies_size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++)
    zap_birdies_kernel<<<calc[ii].blocks,max_threads>>>(fseries,d_birdies,d_widths,bin_width,birdies_size,fseries_size);
  ErrorChecker::check_cuda_error("Error from device_zap_birdies");
  return;
}

//--------------coincidence matching--------------//

__global__ 
void coincidence_kernel(float** arrays, float* out_array,
			int narrays, size_t size,
			float thresh, int beam_thresh)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int count = 0;
  for (int ii=0;ii<narrays;ii++)
    if (arrays[ii][idx]>thresh)
      count++;
  out_array[idx] = (count<beam_thresh);
}

void device_coincidencer(float** arrays, float* out_array, 
			 int narrays, size_t size,
			 float thresh, int beam_thresh,
			 unsigned int max_blocks, 
			 unsigned int max_threads)
{
  
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++)
    coincidence_kernel<<<calc[ii].blocks,max_threads>>>
      (arrays,out_array,narrays,size,thresh,beam_thresh);
  ErrorChecker::check_cuda_error("Error from device_coincidencer");
  return;
  
}

//--------Correlation tools--------//

__global__ void conjugate_kernel(hipfftComplex* x, unsigned int size, 
				 unsigned int gulp_idx){
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_idx;
  if (idx<size)
    x[idx].y *= -1.0;
}

void device_conjugate(hipfftComplex* x, unsigned int size,
		      unsigned int max_blocks,
		      unsigned int max_threads)
{
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++)
    conjugate_kernel<<<calc[ii].blocks,max_threads>>>(x,size,calc[ii].data_idx);
  ErrorChecker::check_cuda_error("Error from device_conjugate");
  return;
}

__global__ void cuCmulf_inplace_kernel(hipfftComplex* x, hipfftComplex* y, 
						unsigned int size, unsigned int gulp_idx){
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_idx;
  if (idx<size)
    y[idx] = hipCmulf(x[idx],y[idx]);
}

void device_cuCmulf_inplace(hipfftComplex* x, hipfftComplex* y,
			    unsigned int size,
			    unsigned int max_blocks,
			    unsigned int max_threads)
{
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++)
    cuCmulf_inplace_kernel<<<calc[ii].blocks,max_threads>>>(x,y,size,calc[ii].data_idx);
  ErrorChecker::check_cuda_error("Error from device_cuCmulf_inplace");
  return;
}

//--------type converter--------//
//This is to get around the stupid thrust copy issue

template <class X,class Y> __global__
void conversion_kernel(X* x, Y* y, unsigned int size,
                       unsigned int gulp_idx)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_idx;
  if (idx<size)
    y[idx] = x[idx];
  return;
}

template __global__ void conversion_kernel<char,float>(char*,float*,unsigned int,unsigned int);
template __global__ void conversion_kernel<unsigned char,float>(unsigned char*,float*,unsigned int,unsigned int);

template <class X,class Y>
void device_conversion(X* x, Y* y, unsigned int size,
                       unsigned int max_blocks,
                       unsigned int max_threads)
{
  BlockCalculator calc(size, max_blocks, max_threads);
  for (int ii=0;ii<calc.size();ii++)
    conversion_kernel<X,Y> <<<calc[ii].blocks,max_threads>>>(x,y,size,calc[ii].data_idx);
  ErrorChecker::check_cuda_error("Error from device_conversion");
  return;
}

template void device_conversion<char,float>(char*, float*, unsigned int, unsigned int, unsigned int);
template void device_conversion<unsigned char,float>(unsigned char*, float*, unsigned int, unsigned int, unsigned int);
template void device_conversion<unsigned int,float>(unsigned int*, float*, unsigned int, unsigned int, unsigned int);


